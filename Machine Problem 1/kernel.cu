// Liam Salass
// 20229595

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

int cores(int major, int minor) {
	//Define GPU arch types using SM version to determine # cores
	

	switch (major) {
	case 2:
		if (minor == 1) return 48;
		else return 32;
		break;
	case 3:
		return 192;
		break;
	case 5:
		return 128;
		break;
	case 6:
		if ((minor == 1) || (minor == 2)) return 128;
		else if (minor == 0) return 64;
		break;
	case 7:
		if ((minor == 0) || (minor == 5)) return 64;
		break;
	}
	printf("Failed to find # cores for Major = %d and Minor = %d \n", major, minor);
	printf("Returned -1\n");
	return -1;
}


int main()
{
	int nDev;

	//Get count of devices
	hipGetDeviceCount(&nDev);

	if (nDev == 0){
		printf("No devices");
	}
	else {
		for (int i = 0; i < nDev; i++) {
			//Get information about each cuda dev
			hipDeviceProp_t dp;
			hipGetDeviceProperties(&dp, i);

			//Print information
			printf("Device Number : %d\n", i);
			printf("\tDevice Name: %s\n", dp.name);
			printf("\tClock Rate: %d\n", dp.clockRate);
			printf("\t# Multiprocessors: %d\n", dp.multiProcessorCount);
			printf("\t# CUDA cores: %d\n", dp.multiProcessorCount * cores(dp.major, dp.minor)); //Multiply number of processors by number of cores
			printf("\tWarp Size: %d\n", dp.warpSize);
			printf("\tGlobal Memory: %ld\n", dp.totalGlobalMem);
			printf("\tConstant Memory: %ld\n", dp.totalConstMem);
			printf("\tShared Memory per block: %ld\n", dp.sharedMemPerBlock);
			printf("\tRegister per block: %d\n", dp.regsPerBlock);
			printf("\tMax threads per block: %d\n", dp.maxThreadsPerBlock);
			printf("\tMax block dimensions: (%d, %d, %d)\n", 
				dp.maxThreadsDim[0],
				dp.maxThreadsDim[1],
				dp.maxThreadsDim[2]);
			printf("\tMax Grid dimensions: (%d, %d, %d)\n",
				dp.maxGridSize[0],
				dp.maxGridSize[1],
				dp.maxGridSize[2]);

		}
	}

}
